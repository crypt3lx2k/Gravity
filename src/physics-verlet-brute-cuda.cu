#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C" {
#include "physics.h"
}

#define BLOCK_SIZE 512

static const value G = GRAVITATIONAL_CONSTANT;
static const value E = SOFTENING*SOFTENING;

static int memory_loaded;

static value * a0x;
static value * a0y;

static value * a1x;
static value * a1y;

static value * dpx;
static value * dpy;

static value * dvx;
static value * dvy;

static value * dm;

static inline void physics_swap (void) {
  value * tx;
  value * ty;

  tx = a0x;
  a0x = a1x;
  a1x = tx;

  ty = a0y;
  a0y = a1y;
  a1y = ty;
}

static inline void physics_load_memory (size_t n,
					const value * px,
					const value * py,
					const value * vx,
					const value * vy,
					const value * m) {
  if (memory_loaded)
    return;

  hipMemcpy(dpx, px, n*sizeof(value), hipMemcpyHostToDevice);
  hipMemcpy(dpy, py, n*sizeof(value), hipMemcpyHostToDevice);

  hipMemcpy(dvx, vx, n*sizeof(value), hipMemcpyHostToDevice);
  hipMemcpy(dvy, vy, n*sizeof(value), hipMemcpyHostToDevice);

  hipMemcpy(dm,  m,  n*sizeof(value), hipMemcpyHostToDevice);

  memory_loaded = 1;
}

static inline void physics_offload_memory (size_t n,
					   value * px, value * py,
					   value * vx, value * vy) {
  hipMemcpy(px, dpx, n*sizeof(value), hipMemcpyDeviceToHost);
  hipMemcpy(py, dpy, n*sizeof(value), hipMemcpyDeviceToHost);

  hipMemcpy(vx, dvx, n*sizeof(value), hipMemcpyDeviceToHost);
  hipMemcpy(vy, dvy, n*sizeof(value), hipMemcpyDeviceToHost);
}

__device__
void physics_advance_positions_inner (value dt, int n,
				      const value * v,
				      const value * a0,
				      value * p) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i >= n)
    return;

  p[i] += (v[i] + value_literal(0.5)*a0[i]*dt)*dt;
}

__global__
void physics_advance_positions (value dt, int n,
				const value *  vx, const value *  vy,
				const value * a0x, const value * a0y,
				value * px, value * py) {
  physics_advance_positions_inner(dt, n, vx, a0x, px);
  physics_advance_positions_inner(dt, n, vy, a0y, py);
}

__device__
value2 physics_calculate_forces_inner (int n, value2 pi, value3 pj, value2 ai) {
  value2 r;
  value  s;

  r.x = pj.x - pi.x;
  r.y = pj.y - pi.y;

  s = (r.x*r.x + r.y*r.y) + E;
  s = s*s*s;
  s = rsqrtv(s);

  s = G*s*pj.z;

  ai.x += r.x*s;
  ai.y += r.y*s;

  return ai;
}

__global__
void physics_calculate_forces (int n,
			       const value * px, const value * py,
			       const value * m,
			       value * a1x, value * a1y) {
  extern __shared__ value3 sp[];

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int tile;

  if (i >= n)
    return;

  value2 pi = {px[i], py[i]};
  value2 ai = {value_literal(0.0), value_literal(0.0)};

  for (tile = 0; tile < gridDim.x; tile++) {
    int tile_j = tile*blockDim.x + threadIdx.x;
    int j;

    if (tile_j < n) {
      sp[threadIdx.x].x = px[tile_j];
      sp[threadIdx.x].y = py[tile_j];
      sp[threadIdx.x].z =  m[tile_j];
    } else {
      sp[threadIdx.x].x = value_literal(0.0);
      sp[threadIdx.x].y = value_literal(0.0);
      sp[threadIdx.x].z = value_literal(0.0);
    }
    __syncthreads();

#pragma unroll 64
    for (j = 0; j < blockDim.x; j++)
      ai = physics_calculate_forces_inner(n, pi, sp[j], ai);
    __syncthreads();
  }

  a1x[i] = ai.x;
  a1y[i] = ai.y;
}

__device__
void physics_advance_velocities_inner (value dt, int n,
				       const value * a0,
				       const value * a1,
				       value * v) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i >= n)
    return;

  v[i] += value_literal(0.5)*(a0[i]+a1[i])*dt;
}

__global__
void physics_advance_velocities (value dt, int n,
				 const value * a0x, const value * a0y,
				 const value * a1x, const value * a1y,
				 value * vx, value * vy) {
  physics_advance_velocities_inner(dt, n, a0x, a1x, vx);
  physics_advance_velocities_inner(dt, n, a0y, a1y, vy);
}

void physics_advance (value dt, size_t n,
		      value * px, value * py,
		      value * vx, value * vy,
		      value * m) {
  int blockSize  = BLOCK_SIZE;
  int gridSize   = (n + blockSize-1)/blockSize;
  int sharedSize = blockSize*3*sizeof(value);

  physics_load_memory(n, px, py, vx, vy, m);  

  physics_advance_positions<<<gridSize, blockSize>>>(dt, n, dvx, dvy, a0x, a0y, dpx, dpy);
  physics_calculate_forces<<<gridSize, blockSize, sharedSize>>>(n, dpx, dpy, dm, a1x, a1y);
  physics_advance_velocities<<<gridSize, blockSize>>>(dt, n, a0x, a0y, a1x, a1y, dvx, dvy);

  physics_swap();

  physics_offload_memory(n, px, py, vx, vy);
}

void physics_free (void) {
  hipFree(a0x);
  hipFree(a0y);

  hipFree(a1x);
  hipFree(a1y);

  hipFree(dpx);
  hipFree(dpy);

  hipFree(dvx);
  hipFree(dvy);

  hipFree(dm);

  hipDeviceReset();
}

void physics_init (size_t n) {
  hipMalloc(&a0x, n*sizeof(value));
  hipMalloc(&a0y, n*sizeof(value));

  hipMalloc(&a1x, n*sizeof(value));
  hipMalloc(&a1y, n*sizeof(value));

  hipMalloc(&dpx, n*sizeof(value));
  hipMalloc(&dpy, n*sizeof(value));

  hipMalloc(&dvx, n*sizeof(value));
  hipMalloc(&dvy, n*sizeof(value));

  hipMalloc(&dm, n*sizeof(value));
}

void physics_reset (size_t n) {
  hipMemset(a0x, 0, n*sizeof(value));
  hipMemset(a0y, 0, n*sizeof(value));
  hipMemset(a1x, 0, n*sizeof(value));
  hipMemset(a1y, 0, n*sizeof(value));

  memory_loaded = 0;
}
