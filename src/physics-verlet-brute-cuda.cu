#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C" {
#include "physics.h"
}

#define value1 float1
#define value2 float2
#define value3 float3
#define value4 float4

#define rsqrtv(x) rsqrtf((x))

static int memory_loaded;

static value * a0x;
static value * a0y;

static value * a1x;
static value * a1y;

static value * dpx;
static value * dpy;

static value * dvx;
static value * dvy;

static value * dm;

static inline void physics_swap (void) {
  value * tx;
  value * ty;

  tx = a0x;
  a0x = a1x;
  a1x = tx;

  ty = a0y;
  a0y = a1y;
  a1y = ty;
}

static inline void physics_load_memory (size_t n,
					const value * px,
					const value * py,
					const value * vx,
					const value * vy,
					const value * m) {
  hipMemcpy(dpx, px, n*sizeof(value), hipMemcpyHostToDevice);
  hipMemcpy(dpy, py, n*sizeof(value), hipMemcpyHostToDevice);

  hipMemcpy(dvx, vx, n*sizeof(value), hipMemcpyHostToDevice);
  hipMemcpy(dvy, vy, n*sizeof(value), hipMemcpyHostToDevice);

  hipMemcpy(dm,  m,  n*sizeof(value), hipMemcpyHostToDevice);

  memory_loaded = 1;
}

static inline void physics_offload_memory (size_t n,
					   value * px, value * py,
					   value * vx, value * vy) {
  hipMemcpy(px, dpx, n*sizeof(value), hipMemcpyDeviceToHost);
  hipMemcpy(py, dpy, n*sizeof(value), hipMemcpyDeviceToHost);

  hipMemcpy(vx, dvx, n*sizeof(value), hipMemcpyDeviceToHost);
  hipMemcpy(vy, dvy, n*sizeof(value), hipMemcpyDeviceToHost);
}

__device__
void physics_advance_update_position (value dt, int n,
				      const value * v,
				      const value * a0,
				      value * p) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i >= n)
    return;

  p[i] += (v[i] + value_literal(0.5)*a0[i]*dt)*dt;
}

__device__
value2 physics_advance_calculate_inner (value2 pi,
					value3 pj,
					value2 ai) {
  value2 r;
  value s;

  r.x = pj.x - pi.x;
  r.y = pj.y - pi.y;

  s = (r.x*r.x + r.y*r.y) + SOFTENING*SOFTENING;
  s = s*s*s;
  s = rsqrtv(s);

  s *= GRAVITATIONAL_CONSTANT*pj.z;

  ai.x += r.x*s;
  ai.y += r.y*s;

  return ai;
}

__device__
void physics_advance_calculate_forces (int n,
				       const value * px,
				       const value * py,
				       const value * m,
				       value * a1x,
				       value * a1y) {
  extern __shared__ value3 shared_storage[];

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int tile;

  if (i >= n)
    return;

  value2 ai = {value_literal(0.0), value_literal(0.0)};
  value2 pi;

  pi.x = px[i];
  pi.y = py[i];

  for (tile = 0; tile*blockDim.x < n; tile++) {
    int j;
    int tile_j = tile*blockDim.x + threadIdx.x;

    if (tile_j < n) {
      shared_storage[threadIdx.x].x = px[tile_j];
      shared_storage[threadIdx.x].y = py[tile_j];
      shared_storage[threadIdx.x].z = m[tile_j];
    }
    __syncthreads();

#pragma unroll 64
    for (j = 0; j < blockDim.x; j++) {
      ai = physics_advance_calculate_inner(pi, shared_storage[j], ai);
    }
    __syncthreads();
  }

  a1x[i] = ai.x;
  a1y[i] = ai.y;
}

__device__
void physics_advance_update_velocity (value dt, int n,
				      const value * a0,
				      const value * a1,
				      value * v) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if (i >= n)
    return;

  v[i] += value_literal(0.5)*(a0[i]+a1[i])*dt;
}

__global__
void physics_advance_kernel (value dt, int n,
			     value * px, value * py,
			     value * vx, value * vy,
			     value * m,
			     value * a0x, value * a0y,
			     value * a1x, value * a1y) {
  physics_advance_update_position(dt, n, vx, a0x, px);
  physics_advance_update_position(dt, n, vy, a0y, py);
  __syncthreads();

  physics_advance_calculate_forces(n, px, py, m, a1x, a1y);
  __syncthreads();

  physics_advance_update_velocity(dt, n, a0x, a1x, vx);
  physics_advance_update_velocity(dt, n, a0y, a1y, vy);
}

void physics_advance (value dt, size_t n,
		      value * px, value * py,
		      value * vx, value * vy,
		      value * m) {
  int blockSize  = 512;
  int gridSize   = (n + blockSize-1)/blockSize;
  int sharedSize = blockSize*3*sizeof(value);

  if (!memory_loaded)
    physics_load_memory(n, px, py, vx, vy, m);

  physics_advance_kernel<<<gridSize, blockSize, sharedSize>>>(dt, n,
							      dpx, dpy,
							      dvx, dvy,
							      dm,
							      a0x, a0y,
							      a1x, a1y);
  physics_swap();

  physics_offload_memory(n, px, py, vx, vy);
}

void physics_free (void) {
  hipFree(a0x);
  hipFree(a0y);

  hipFree(a1x);
  hipFree(a1y);

  hipFree(dpx);
  hipFree(dpy);

  hipFree(dvx);
  hipFree(dvy);

  hipFree(dm);

  hipDeviceReset();
}

void physics_init (size_t n) {
  hipMalloc(&a0x, n*sizeof(value));
  hipMalloc(&a0y, n*sizeof(value));

  hipMalloc(&a1x, n*sizeof(value));
  hipMalloc(&a1y, n*sizeof(value));

  hipMalloc(&dpx, n*sizeof(value));
  hipMalloc(&dpy, n*sizeof(value));

  hipMalloc(&dvx, n*sizeof(value));
  hipMalloc(&dvy, n*sizeof(value));

  hipMalloc(&dm, n*sizeof(value));
}

void physics_reset (size_t n) {
  hipMemset(a0x, 0, n*sizeof(value));
  hipMemset(a0y, 0, n*sizeof(value));
  hipMemset(a1x, 0, n*sizeof(value));
  hipMemset(a1y, 0, n*sizeof(value));

  memory_loaded = 0;
}
